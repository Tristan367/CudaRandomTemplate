#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 25

__global__ void init(unsigned int seed, hiprandState_t* states) 
{
    hiprand_init(seed, blockIdx.x, 10000, &states[blockIdx.x]); // initializing the random state for each thread
}

__global__ void randoms(hiprandState_t* states, unsigned int* numbers) 
{
    numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % 100;
}

int main() 
{
    // initializing the random states for each thread
    hiprandState_t* states;
    hipMalloc((void**)&states, N * sizeof(hiprandState_t)); // allocate space on the GPU for the random states 
    //init << <N, 1 >> > (time(0), states); // seed using time
    init << <N, 1 >> > (13, states); // constant seed for debugging

    // send a brain to the GPU to mutate each neuron in parrelell

    unsigned int* gpu_nums;
    hipMalloc((void**)&gpu_nums, N * sizeof(unsigned int));

    randoms << <N, 1 >> > (states, gpu_nums);

    unsigned int cpu_nums[N];
    hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost); // copy the random numbers back 

    for (int i = 0; i < N; i++) {
        printf("%u\n", cpu_nums[i]);
    }

    hipFree(states);
    hipFree(gpu_nums);
    return 0;
}
